#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <cstdint>
#include <hip/hip_runtime.h>

typedef double d_t;

// 3D点结构体
struct d3_t {
    d_t x, y, z;
};

// 计算3D向量的欧几里得范数
__host__ __device__
inline d_t norm(d_t dx, d_t dy, d_t dz) {
    return sqrt(dx * dx + dy * dy + dz * dz);
}

// 预处理核函数：计算反射镜点到光源的距离
__global__
void precomputeMirDistances(const d_t* __restrict__ mir_x,
                            const d_t* __restrict__ mir_y,
                            const d_t* __restrict__ mir_z,
                            int64_t mirn,
                            d3_t src,
                            d_t* __restrict__ distances) {
    const int64_t idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < mirn) {
        const d_t dx = mir_x[idx] - src.x;
        const d_t dy = mir_y[idx] - src.y;
        const d_t dz = mir_z[idx] - src.z;
        distances[idx] = norm(dx, dy, dz);
    }
}

// 主核函数：计算传感器点的 Iv 值
__global__
void computeIvKernel(const d_t* __restrict__ mir_x,
                     const d_t* __restrict__ mir_y,
                     const d_t* __restrict__ mir_z,
                     const d_t* __restrict__ mir_dist,
                     int64_t mirn,
                     const d_t* __restrict__ sen_x,
                     const d_t* __restrict__ sen_y,
                     const d_t* __restrict__ sen_z,
                     int64_t senn,
                     d_t* __restrict__ data) {
    extern __shared__ d_t smem[];
    d_t* real_sums = smem;
    d_t* imag_sums = smem + blockDim.x;

    const int sen_idx = blockIdx.x;
    if (sen_idx >= senn) return;

    const d_t sv_x = sen_x[sen_idx];
    const d_t sv_y = sen_y[sen_idx];
    const d_t sv_z = sen_z[sen_idx];

    const int tid = threadIdx.x;
    const int nthreads = blockDim.x;

    d_t sum_real = 0.0, sum_imag = 0.0;
    const d_t factor = 2.0 * M_PI / 500e-6;

    for (int64_t mu = tid; mu < mirn; mu += nthreads) {
        const d_t dx = mir_x[mu] - sv_x;
        const d_t dy = mir_y[mu] - sv_y;
        const d_t dz = mir_z[mu] - sv_z;
        const d_t dist = mir_dist[mu] + norm(dx, dy, dz);
        const d_t angle = dist * factor;

        sum_real += cos(angle);
        sum_imag += sin(angle);
    }

    real_sums[tid] = sum_real;
    imag_sums[tid] = sum_imag;
    __syncthreads();

    // 树状归约
    for (int s = nthreads / 2; s > 0; s >>= 1) {
        if (tid < s) {
            real_sums[tid] += real_sums[tid + s];
            imag_sums[tid] += imag_sums[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        const d_t total_real = real_sums[0];
        const d_t total_imag = imag_sums[0];
        data[sen_idx] = sqrt(total_real * total_real + total_imag * total_imag);
    }
}

int main() {
    // 打开输入文件
    FILE* fi = fopen("in.data", "rb");
    if (!fi) {
        perror("Failed to open input file");
        return EXIT_FAILURE;
    }

    // 读取点光源坐标
    d3_t src;
    fread(&src, sizeof(d3_t), 1, fi);

    // 读取反射镜点数量和坐标
    int64_t mirn;
    fread(&mirn, sizeof(int64_t), 1, fi);
    d_t* mir_x = (d_t*)malloc(mirn * sizeof(d_t));
    d_t* mir_y = (d_t*)malloc(mirn * sizeof(d_t));
    d_t* mir_z = (d_t*)malloc(mirn * sizeof(d_t));
    for (int64_t i = 0; i < mirn; i++) {
        d3_t point;
        fread(&point, sizeof(d3_t), 1, fi);
        mir_x[i] = point.x;
        mir_y[i] = point.y;
        mir_z[i] = point.z;
    }

    // 读取传感器点数量和坐标
    int64_t senn;
    fread(&senn, sizeof(int64_t), 1, fi);
    d_t* sen_x = (d_t*)malloc(senn * sizeof(d_t));
    d_t* sen_y = (d_t*)malloc(senn * sizeof(d_t));
    d_t* sen_z = (d_t*)malloc(senn * sizeof(d_t));
    for (int64_t i = 0; i < senn; i++) {
        d3_t point;
        fread(&point, sizeof(d3_t), 1, fi);
        sen_x[i] = point.x;
        sen_y[i] = point.y;
        sen_z[i] = point.z;
    }

    fclose(fi);

    // 分配主机端结果存储
    d_t* data = (d_t*)malloc(senn * sizeof(d_t));

    // 分配设备端内存
    d_t *d_mir_x, *d_mir_y, *d_mir_z, *d_mir_dist;
    d_t *d_sen_x, *d_sen_y, *d_sen_z, *d_data;
    hipMalloc(&d_mir_x, mirn * sizeof(d_t));
    hipMalloc(&d_mir_y, mirn * sizeof(d_t));
    hipMalloc(&d_mir_z, mirn * sizeof(d_t));
    hipMalloc(&d_mir_dist, mirn * sizeof(d_t));
    hipMalloc(&d_sen_x, senn * sizeof(d_t));
    hipMalloc(&d_sen_y, senn * sizeof(d_t));
    hipMalloc(&d_sen_z, senn * sizeof(d_t));
    hipMalloc(&d_data, senn * sizeof(d_t));

    // 将数据拷贝到设备端
    hipMemcpy(d_mir_x, mir_x, mirn * sizeof(d_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mir_y, mir_y, mirn * sizeof(d_t), hipMemcpyHostToDevice);
    hipMemcpy(d_mir_z, mir_z, mirn * sizeof(d_t), hipMemcpyHostToDevice);
    hipMemcpy(d_sen_x, sen_x, senn * sizeof(d_t), hipMemcpyHostToDevice);
    hipMemcpy(d_sen_y, sen_y, senn * sizeof(d_t), hipMemcpyHostToDevice);
    hipMemcpy(d_sen_z, sen_z, senn * sizeof(d_t), hipMemcpyHostToDevice);

    // 预处理反射镜点到光源的距离
    const int BLOCK_SIZE = 256;
    precomputeMirDistances<<<(mirn + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(
        d_mir_x, d_mir_y, d_mir_z, mirn, src, d_mir_dist
    );

    // 主计算核函数
    size_t shared_mem = 2 * BLOCK_SIZE * sizeof(d_t);
    computeIvKernel<<<senn, BLOCK_SIZE, shared_mem>>>(
        d_mir_x, d_mir_y, d_mir_z, d_mir_dist, mirn,
        d_sen_x, d_sen_y, d_sen_z, senn, d_data
    );

    // 将结果拷贝回主机端
    hipMemcpy(data, d_data, senn * sizeof(d_t), hipMemcpyDeviceToHost);

    // 将结果写入输出文件
    FILE* fo = fopen("out.data", "wb");
    if (!fo) {
        perror("Failed to open output file");
        free(mir_x); free(mir_y); free(mir_z);
        free(sen_x); free(sen_y); free(sen_z);
        free(data);
        hipFree(d_mir_x); hipFree(d_mir_y); hipFree(d_mir_z); hipFree(d_mir_dist);
        hipFree(d_sen_x); hipFree(d_sen_y); hipFree(d_sen_z); hipFree(d_data);
        return EXIT_FAILURE;
    }
    fwrite(data, sizeof(d_t), senn, fo);
    fclose(fo);

    // 释放主机内存
    free(mir_x); free(mir_y); free(mir_z);
    free(sen_x); free(sen_y); free(sen_z);
    free(data);

    // 释放设备内存
    hipFree(d_mir_x); hipFree(d_mir_y); hipFree(d_mir_z); hipFree(d_mir_dist);
    hipFree(d_sen_x); hipFree(d_sen_y); hipFree(d_sen_z); hipFree(d_data);

    return 0;
}