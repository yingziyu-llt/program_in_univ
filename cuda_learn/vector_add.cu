
#include <hip/hip_runtime.h>
#include<stdio.h>
const int N = 1000;

static void HandleError( hipError_t err,
                         const char *file,
                         int line ) {
    if (err != hipSuccess) {
        printf( "%s in %s at line %d\n", hipGetErrorString( err ),
                file, line );
        exit( EXIT_FAILURE );
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


__global__ void add(int *a,int *b,int *c)
{
	int xx = blockIdx.x;
	if(xx < N)
	{
		c[xx] = a[xx] + b[xx];
	}
}

int main()
{
	int a[10000],b[10000];
	int c[10000];
	for(int i = 0;i < N;i++)
		a[i] = i;
	for(int i = 0;i < N;i++)
		b[i] = i + 1;
	int *dev_a,*dev_b,*dev_c;
	HANDLE_ERROR(hipMalloc((void**)&dev_a,N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_b,N * sizeof(int)));
	HANDLE_ERROR(hipMalloc((void**)&dev_c,N * sizeof(int)));
	HANDLE_ERROR(hipMemcpy(dev_a,a,N * sizeof(int),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_b,b,N * sizeof(int),hipMemcpyHostToDevice));
	HANDLE_ERROR(hipMemcpy(dev_c,c,N * sizeof(int),hipMemcpyHostToDevice));
	add<<<N,1>>>(dev_a,dev_b,dev_c);
	HANDLE_ERROR(hipMemcpy(c,dev_c,N * sizeof(int),hipMemcpyDeviceToHost));
	for(int i = 0;i < N;i++)
		printf("%d\n",c[i]);
	hipFree(dev_a);hipFree(dev_b);hipFree(dev_c);
}
