/*
 * Copyright (C) 2024 Letian Lin <yingziyu-Lin@outlook.com>
 *
 * Distributed under terms of the GNU AGPLv3+ license.
 * 
 * Run for our better future!
 */


#include <hip/hip_runtime.h>
#include<stdio.h>

__global__ void add(int a,int b,int *c)
{
	*c = a + b;
}

int main()
{
	int c;
	int *dev_c;
	hipMalloc((void**)&dev_c,sizeof(int));
	add<<<1,1>>>(2,7,dev_c);
	hipMemcpy(&c,dev_c,sizeof(int),hipMemcpyDeviceToHost);
	printf("%d",c);
	hipFree(dev_c);
}

