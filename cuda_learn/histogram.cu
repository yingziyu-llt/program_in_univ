#include <iostream>
#include <hip/hip_runtime.h>
#include <random>
#include <ctime>

constexpr int block_size = 256;
constexpr int unroll_factor = 4;  // 每个线程处理4个char

__global__ void histogram(const unsigned int* source, int* hist, int dataSize) {
    // 每个线程处理unroll_factor个char
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * unroll_factor;

    // 局部直方图，减少全局内存原子操作
    __shared__ int local_hist[256];
    for (int i = threadIdx.x; i < 256; i += blockDim.x) {
        local_hist[i] = 0;
    }
    __syncthreads();

    // 处理数据
    if (idx < dataSize) {
        unsigned int value_u32 = source[idx / unroll_factor];  // 读取一个int（4个char）
        for (int i = 0; i < unroll_factor; i++) {
            unsigned char charValue = (value_u32 >> (8 * i)) & 0xFF;  // 提取每个char
            atomicAdd(&local_hist[charValue], 1);  // 更新局部直方图
        }
    }
    __syncthreads();

    // 将局部直方图合并到全局直方图
    for (int i = threadIdx.x; i < 256; i += blockDim.x) {
        atomicAdd(&hist[i], local_hist[i]);
    }
}

int main() {
    const int dataSize = 100000000;  // 数据大小
    char* source = new char[dataSize];
    int* hist = new int[256];

    // 初始化数据
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_int_distribution<> dis(0, 255);
    for (int i = 0; i < dataSize; i++) {
        source[i] = static_cast<char>(dis(gen));
    }

    // 初始化直方图
    for (int i = 0; i < 256; i++) {
        hist[i] = 0;
    }

    // 设备端内存分配
    unsigned int* d_source;
    int* d_hist;
    hipMalloc((void**)&d_source, (dataSize / unroll_factor) * sizeof(unsigned int));
    hipMalloc((void**)&d_hist, 256 * sizeof(int));

    // 将数据从主机拷贝到设备
    hipMemcpy(d_source, reinterpret_cast<unsigned int*>(source), (dataSize / unroll_factor) * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(d_hist, hist, 256 * sizeof(int), hipMemcpyHostToDevice);

    // 启动核函数
    int grid_size = (dataSize / unroll_factor + block_size - 1) / block_size;
    clock_t time_start = clock();
    histogram<<<grid_size, block_size>>>(d_source, d_hist, dataSize);
    hipDeviceSynchronize();
    clock_t time_end = clock();

    // 将直方图从设备拷贝回主机
    hipMemcpy(hist, d_hist, 256 * sizeof(int), hipMemcpyDeviceToHost);

    // 打印耗时
    std::cout << "Time: " << (time_end - time_start) * 1000 / CLOCKS_PER_SEC << " ms" << std::endl;

    // 释放设备内存
    hipFree(d_source);
    hipFree(d_hist);

    // 释放主机内存
    delete[] source;
    delete[] hist;

    return 0;
}